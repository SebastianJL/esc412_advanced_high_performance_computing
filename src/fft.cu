#include <fstream>
#include <typeinfo>
#include <complex>
#include <stdlib.h>
#include "hipfft/hipfft.h"
#include "blitz/array.h"

#include "aweights.h"
#include "tipsy.h"

using namespace std;
using namespace blitz;

typedef double real_type;
typedef std::complex<real_type> complex_type;
typedef blitz::Array<real_type,3> array3D_r;
typedef blitz::Array<complex_type,3> array3D_c;


//**********************************************************************

void compute_fft_2D_R2C(array3D_r &grid, int N, int Nx) {
    int n[] = {N,N};       // 2D FFT of length NxN
    int inembed[] = {N,2*(N/2+1)};
    int onembed[] = {N,(N/2+1)};
    int howmany = Nx;
    int odist = N*(N/2+1); // Output distance is in "complex"
    int idist = 2*odist;   // Input distance is in "real"
    int istride = 1,       // Elements of each FFT are adjacent
	ostride = 1;

    hipfftHandle plan;
    hipfftPlanMany(&plan,sizeof(n)/sizeof(n[0]), n,
		    inembed,istride,idist,
		    onembed,ostride,odist,
		    HIPFFT_D2Z,howmany);
    hipfftDoubleComplex *data;
    hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*N*N*(N/2+1));
    hipMemcpy(data, grid.dataFirst(), sizeof(hipfftDoubleComplex)*N*N*(N/2+1), hipMemcpyHostToDevice);
    hipfftExecD2Z(plan,reinterpret_cast<hipfftDoubleReal*>(data),data);
    hipMemcpy(grid.dataFirst(), data,sizeof(hipfftDoubleComplex)*N*N*(N/2+1), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(data);
    hipfftDestroy(plan);

}

void compute_fft_1D_C2C(array3D_c &fft_grid, int N, int Nx){
    // 2D FFT of the 1st dimensions: C2C
    int n[] = {N};
    int *inembed = n, *onembed = n;
    int howmany = N*(N/2+1);
    int idist = 1;
    int odist = 1;
    int istride = N*(N/2+1), ostride = N*(N/2+1);

    hipfftHandle plan;
    hipfftPlanMany(&plan,sizeof(n)/sizeof(n[0]), n,
                    inembed,istride,idist,
                    onembed,ostride,odist,
                    HIPFFT_Z2Z,howmany);
    hipfftDoubleComplex *data;
    hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*N*N*(N/2+1));
    hipMemcpy(data, fft_grid.dataFirst(), sizeof(hipfftDoubleComplex)*N*N*(N/2+1), hipMemcpyHostToDevice);
    hipfftExecZ2Z(plan,data,data,HIPFFT_FORWARD);
    hipMemcpy(fft_grid.dataFirst(), data,sizeof(hipfftDoubleComplex)*N*N*(N/2+1), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(data);
    hipfftDestroy(plan);

}

//**********************************************************************

void compute_fft_2D_R2C_stream(array3D_r &grid, array3D_c &fft_grid, int N) {
    int n[] = {N,N};       // 2D FFT of length NxN
    int inembed[] = {N,2*(N/2+1)};
    int onembed[] = {N,(N/2+1)};
    const int howmany = 16;// Number of slabs to do at once
    int odist = N*(N/2+1); // Output distance is in "complex"
    int idist = 2*odist;   // Input distance is in "real"
    int istride = 1,       // Elements of each FFT are adjacent
	ostride = 1;
    const int nStreams = 4;

    // Allocate the CUDA streams. Each stream can execute independently
    hipStream_t stream[nStreams];
    for(auto i=0; i<nStreams; ++i) hipStreamCreate(stream+i);

    // Allocate a single chunk on the GPU, but separate it into blocks
    hipfftDoubleComplex *data[nStreams];
    int block_count = howmany*N*(N/2+1);
    hipMalloc((void**)data, sizeof(hipfftDoubleComplex)*nStreams*block_count);
    for(auto i=1; i<nStreams; ++i) data[i] = data[0] + i*block_count;

    // Create a plan to do "howmany" slabs at a time
    // This plan will be run simultaneously on multiple streams so that
    // means we have to create "work areas" for each. Instead of calling
    // hipfftPlanMany we call hipfftMakePlanMany so we can disable auto allocation
    // of the work area. Later we need to call hipfftSetWorkArea.
    hipfftHandle plan;
    size_t workSize;
    hipfftCreate(&plan);
    hipfftSetAutoAllocation(plan,0);
    hipfftMakePlanMany(plan,sizeof(n)/sizeof(n[0]), n,
		    inembed,istride,idist,
		    onembed,ostride,odist,
		    HIPFFT_D2Z,howmany,&workSize);
    void *workArea[nStreams];
    // We allocate "nStreams" work areas and set pointer to them for each stream
    hipMalloc(&workArea[0],workSize*nStreams);
    for(auto i=1; i<nStreams; ++i) workArea[i] = reinterpret_cast<char*>(workArea[0]) + i*workSize;


    // Distribute the work on the streams
    int iStream = 0;
    for(auto i=grid.lbound(0); i<=grid.ubound(0); i+=howmany) {
	hipMemcpyAsync(data[iStream], &grid(i,0,0), sizeof(hipfftDoubleComplex)*block_count, hipMemcpyHostToDevice,stream[iStream]);
	hipfftSetStream(plan,stream[iStream]);
	hipfftSetWorkArea(plan,workArea[iStream]);
	hipfftExecD2Z(plan,reinterpret_cast<hipfftDoubleReal*>(data[iStream]),data[iStream]);
	hipMemcpyAsync(&grid(i,0,0),data[iStream],sizeof(hipfftDoubleComplex)*block_count, hipMemcpyDeviceToHost,stream[iStream]);
	if (++iStream == nStreams) iStream = 0;
    }
    hipDeviceSynchronize(); // Wait for all streams to complete
    hipFree(data[0]);
    hipFree(workArea[0]);
    hipfftDestroy(plan);
    for(auto i=0; i<nStreams; ++i) hipStreamDestroy(stream[i]);
}



//**********************************************************************
