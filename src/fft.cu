#include <fstream>
#include <typeinfo>
#include <complex>
#include <stdlib.h>
#include "hipfft/hipfft.h"
#include "blitz/array.h"

#include "aweights.h"
#include "tipsy.h"

using namespace std;
using namespace blitz;

typedef double real_type;
typedef std::complex<real_type> complex_type;
typedef blitz::Array<real_type,3> array3D_r;
typedef blitz::Array<complex_type,3> array3D_c;


//**********************************************************************

void compute_fft_2D_R2C(array3D_r &grid, int N, int local_n) {
    int n[] = {N,N};       // 2D FFT of length NxN
    int inembed[] = {N,2*(N/2+1)};
    int onembed[] = {N,(N/2+1)};
    int howmany = local_n;
    int odist = N*(N/2+1); // Output distance is in "complex"
    int idist = 2*odist;   // Input distance is in "real"
    int istride = 1;       // Elements of each FFT are adjacent
	int ostride = 1;

    hipfftHandle plan;
    hipfftPlanMany(&plan,sizeof(n)/sizeof(n[0]), n,
		    inembed,istride,idist,
		    onembed,ostride,odist,
		    HIPFFT_D2Z,howmany);
    hipfftDoubleComplex *data;
    auto data_size = sizeof(hipfftDoubleComplex)*local_n*N*(N/2+1);
    hipMalloc((void**)&data, data_size);
    hipMemcpy(data, grid.dataFirst(), data_size, hipMemcpyHostToDevice);
    hipfftExecD2Z(plan,reinterpret_cast<hipfftDoubleReal*>(data),data);
    hipMemcpy(grid.dataFirst(), data, data_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(data);
    hipfftDestroy(plan);

}

/** 1D complex to complex fft along the second dimension.
 *
 * In the complex-space the grid is N*N*(N/2+1). Since the code is parallelized
 * along the first dimension the actual grid size is local_n*N*(N/2+1).
 * It is assumed that compute_fft_2D_R2C and transpose() have already been
 * called on the grid.
 *
 * @param fft_grid
 * @param N Grid parameter.
 * @param local_n Local size of the first dimension.
 */
void compute_fft_1D_C2C(array3D_c &fft_grid, int N, int local_n){
    int n[] = {N};
    int *inembed = n;  // Effectively ignored. See doc for outermost dimension.
    int *onembed = n;  // Effectively ignored.
    int howmany = N;
    int idist = 1;
    int odist = 1;
    int istride = (N/2+1);
    int ostride = (N/2+1);

    
    for (int slab=0; slab<local_n; slab++) {

        hipfftHandle plan;
        hipfftPlanMany(&plan,sizeof(n)/sizeof(n[0]), n,
                      inembed,istride,idist,
                      onembed,ostride,odist,
                      HIPFFT_Z2Z,howmany);
        
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to plan\n");
        }

        hipfftDoubleComplex *data;
        auto data_size = sizeof(hipfftDoubleComplex)*N*(N/2+1);
        hipMalloc((void**)&data, data_size);

        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");
        }

        int index = slab * N*(N/2 + 1);

        hipMemcpy(data, fft_grid.dataFirst() + index, data_size, hipMemcpyHostToDevice);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to copy to device\n");
        }
        
        hipfftExecZ2Z(plan, data, data, HIPFFT_FORWARD);
        
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to execute plan\n");
        }

        hipMemcpy(fft_grid.dataFirst() + index, data, data_size, hipMemcpyDeviceToHost);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to copyt to host\n");
        }

        hipDeviceSynchronize();
        hipFree(data);
        hipfftDestroy(plan);
    }

}

//void compute_fft_1D_C2C(array3D_c &fft_grid, int N, int local_n){
//    int n[] = {N};
//    int *inembed = n, *onembed = n;
//    int howmany = N*(N/2+1);
//    int idist = 1;
//    int odist = 1;
//    int istride = N*(N/2+1);
//    int ostride = N*(N/2+1);
//
//    hipfftHandle plan;
//    hipfftPlanMany(&plan,sizeof(n)/sizeof(n[0]), n,
//                    inembed,istride,idist,
//                    onembed,ostride,odist,
//                    HIPFFT_Z2Z,howmany);
//    hipfftDoubleComplex *data;
//    hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*N*N*(N/2+1));
//    hipMemcpy(data, fft_grid.dataFirst(), sizeof(hipfftDoubleComplex)*N*N*(N/2+1), hipMemcpyHostToDevice);
//    hipfftExecZ2Z(plan,data,data,HIPFFT_FORWARD);
//    hipMemcpy(fft_grid.dataFirst(), data,sizeof(hipfftDoubleComplex)*N*N*(N/2+1), hipMemcpyDeviceToHost);
//    hipDeviceSynchronize();
//    hipFree(data);
//    hipfftDestroy(plan);
//
//}

//**********************************************************************

void compute_fft_2D_R2C_stream(array3D_r &grid, array3D_c &fft_grid, int N) {
    int n[] = {N,N};       // 2D FFT of length NxN
    int inembed[] = {N,2*(N/2+1)};
    int onembed[] = {N,(N/2+1)};
    const int howmany = 16;// Number of slabs to do at once
    int odist = N*(N/2+1); // Output distance is in "complex"
    int idist = 2*odist;   // Input distance is in "real"
    int istride = 1,       // Elements of each FFT are adjacent
	ostride = 1;
    const int nStreams = 4;

    // Allocate the CUDA streams. Each stream can execute independently
    hipStream_t stream[nStreams];
    for(auto i=0; i<nStreams; ++i) hipStreamCreate(stream+i);

    // Allocate a single chunk on the GPU, but separate it into blocks
    hipfftDoubleComplex *data[nStreams];
    int block_count = howmany*N*(N/2+1);
    hipMalloc((void**)data, sizeof(hipfftDoubleComplex)*nStreams*block_count);
    for(auto i=1; i<nStreams; ++i) data[i] = data[0] + i*block_count;

    // Create a plan to do "howmany" slabs at a time
    // This plan will be run simultaneously on multiple streams so that
    // means we have to create "work areas" for each. Instead of calling
    // hipfftPlanMany we call hipfftMakePlanMany so we can disable auto allocation
    // of the work area. Later we need to call hipfftSetWorkArea.
    hipfftHandle plan;
    size_t workSize;
    hipfftCreate(&plan);
    hipfftSetAutoAllocation(plan,0);
    hipfftMakePlanMany(plan,sizeof(n)/sizeof(n[0]), n,
		    inembed,istride,idist,
		    onembed,ostride,odist,
		    HIPFFT_D2Z,howmany,&workSize);
    void *workArea[nStreams];
    // We allocate "nStreams" work areas and set pointer to them for each stream
    hipMalloc(&workArea[0],workSize*nStreams);
    for(auto i=1; i<nStreams; ++i) workArea[i] = reinterpret_cast<char*>(workArea[0]) + i*workSize;


    // Distribute the work on the streams
    int iStream = 0;
    for(auto i=grid.lbound(0); i<=grid.ubound(0); i+=howmany) {
	hipMemcpyAsync(data[iStream], &grid(i,0,0), sizeof(hipfftDoubleComplex)*block_count, hipMemcpyHostToDevice,stream[iStream]);
	hipfftSetStream(plan,stream[iStream]);
	hipfftSetWorkArea(plan,workArea[iStream]);
	hipfftExecD2Z(plan,reinterpret_cast<hipfftDoubleReal*>(data[iStream]),data[iStream]);
	hipMemcpyAsync(&grid(i,0,0),data[iStream],sizeof(hipfftDoubleComplex)*block_count, hipMemcpyDeviceToHost,stream[iStream]);
	if (++iStream == nStreams) iStream = 0;
    }
    hipDeviceSynchronize(); // Wait for all streams to complete
    hipFree(data[0]);
    hipFree(workArea[0]);
    hipfftDestroy(plan);
    for(auto i=0; i<nStreams; ++i) hipStreamDestroy(stream[i]);
}



//**********************************************************************
